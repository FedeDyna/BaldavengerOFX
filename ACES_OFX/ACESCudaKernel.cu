#include "hip/hip_runtime.h"
#include "ACES_LIB/ACES_IDT.h"
#include "ACES_LIB/ACES_LMT.h"
#include "ACES_LIB/ACES_RRT.h"
#include "ACES_LIB/ACES_ODT.h"
#include "ACES_LIB/ACES_Conversion.h"

__global__ void ACESKernel(const float* p_Input, float* p_Output, int p_Width, int p_Height, 
int p_Direction, int p_IDT, int p_ACESIN, int p_LMT, int p_ACESOUT, int p_RRT, int p_InvRRT, 
int p_ODT, int p_InvODT, float p_Exposure, float p_LMTScale1, float p_LMTScale2, float p_LMTScale3, 
float p_LMTScale4, float p_LMTScale5, float p_LMTScale6, float p_LMTScale7, float p_LMTScale8, 
float p_LMTScale9, float p_LMTScale10, float p_LMTScale11, float p_LMTScale12, float p_LMTScale13)
{
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;

if ((x < p_Width) && (y < p_Height))
{
const int index = ((y * p_Width) + x) * 4;

float3 aces;
aces.x = p_Input[index + 0];
aces.y = p_Input[index + 1];
aces.z = p_Input[index + 2];

if(p_Direction == 0)
{

switch (p_IDT)
{
case 0:
{

}
break;

case 1:
{
aces = IDT_Alexa_v3_logC_EI800(aces);
}
break;

case 2:
{
aces = IDT_Alexa_v3_raw_EI800_CCT65(aces);
}
break;

case 3:
{
aces = ADX10_to_ACES(aces);
}
break;

case 4:
{
aces = ADX16_to_ACES(aces);
}
}

if(p_Exposure != 0.0f)
{
aces.x *= powf(2.0f, p_Exposure);
aces.y *= powf(2.0f, p_Exposure);
aces.z *= powf(2.0f, p_Exposure);
}

switch (p_ACESIN)
{
case 0:
{

}
break;

case 1:
{
aces = ACES_to_ACEScc(aces);
}
break;

case 2:
{
aces = ACES_to_ACEScct(aces);
}
break;

case 3:
{
aces = ACES_to_ACEScg(aces);
}
break;

case 4:
{
aces = ACES_to_ACESproxy10(aces);
}
break;

case 5:
{
aces = ACES_to_ACESproxy12(aces);
}
}

switch (p_LMT)
{
case 0:
{

}
break;

case 1:
{
aces = LMT_Bleach_Bypass(aces);
}
break;

case 2:
{
aces = LMT_PFE(aces);
}
break;

case 3:
{
aces = scale_C(aces, p_LMTScale1);

float3 SLOPE = {p_LMTScale2, p_LMTScale2, p_LMTScale2};
float3 OFFSET = {p_LMTScale3, p_LMTScale3, p_LMTScale3};
float3 POWER = {p_LMTScale4, p_LMTScale4, p_LMTScale4};
float SAT = p_LMTScale5;

aces = ASCCDL_inACEScct(aces, SLOPE, OFFSET, POWER, SAT);
aces = gamma_adjust_linear(aces, p_LMTScale6, p_LMTScale7);
aces = rotate_H_in_H(aces, p_LMTScale8, p_LMTScale9, p_LMTScale10);
aces = scale_C_at_H(aces, p_LMTScale11, p_LMTScale12, p_LMTScale13);
}
}

switch (p_ACESOUT)
{
case 0:
{

}
break;

case 1:
{
aces = ACEScc_to_ACES(aces);
}
break;

case 2:
{
aces = ACEScct_to_ACES(aces);
}
break;

case 3:
{
aces = ACEScg_to_ACES(aces);
}
break;

case 4:
{
aces = ACESproxy10_to_ACES(aces);
}
break;

case 5:
{
aces = ACESproxy12_to_ACES(aces);
}
}

if(p_RRT == 1)
{
aces = RRT(aces);
}

switch (p_ODT)
{
case 0:
{

}
break;

case 1:
{
aces = ODT_Rec709_100nits_dim(aces);
}
break;

case 2:
{
aces = ODT_Rec2020_100nits_dim(aces);
}
break;

case 3:
{
aces = ODT_Rec2020_ST2084_1000nits(aces);
}
break;

case 4:
{
aces = ODT_RGBmonitor_100nits_dim(aces);
}
}

} else {

switch (p_InvODT)
{
case 0:
{

}
break;

case 1:
{
aces = InvODT_Rec709_100nits_dim(aces);
}
break;

case 2:
{
aces = InvODT_Rec2020_100nits_dim(aces);
}
break;

case 3:
{
aces = InvODT_Rec2020_ST2084_1000nits(aces);
}
break;

case 4:
{
aces = InvODT_RGBmonitor_100nits_dim(aces);
}
}

if(p_InvRRT == 1)
{
aces = InvRRT(aces);
}

}

																										
p_Output[index + 0] = aces.x;
p_Output[index + 1] = aces.y;
p_Output[index + 2] = aces.z;
p_Output[index + 3] = p_Input[index + 3];
}
}

void RunCudaKernel(const float* p_Input, float* p_Output, int p_Width, int p_Height, 
int p_Direction, int p_IDT, int p_ACESIN, int p_LMT, int p_ACESOUT, int p_RRT, 
int p_InvRRT, int p_ODT, int p_InvODT, float p_Exposure, float* p_LMTScale)
{
dim3 threads(128, 1, 1);
dim3 blocks(((p_Width + threads.x - 1) / threads.x), p_Height, 1);

ACESKernel<<<blocks, threads>>>(p_Input, p_Output, p_Width, p_Height, p_Direction, p_IDT, p_ACESIN, p_LMT, p_ACESOUT, 
p_RRT, p_InvRRT, p_ODT, p_InvODT, p_Exposure, p_LMTScale[0], p_LMTScale[1], p_LMTScale[2], p_LMTScale[3], p_LMTScale[4], 
p_LMTScale[5], p_LMTScale[6], p_LMTScale[7], p_LMTScale[8], p_LMTScale[9], p_LMTScale[10], p_LMTScale[11], p_LMTScale[12]);

}