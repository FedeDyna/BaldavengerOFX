
#include <hip/hip_runtime.h>
__global__ void FilmGradeKernel(int p_Width, int p_Height, float p_ExpR, float p_ExpG, float p_ExpB, 
    float p_ContR, float p_ContG, float p_ContB, float p_SatR, float p_SatG, float p_SatB, 
    float p_ShadR, float p_ShadG, float p_ShadB, float p_MidR, float p_MidG, float p_MidB, 
    float p_HighR, float p_HighG, float p_HighB, float p_ShadP, float p_HighP, float p_ContP, 
    float p_DisplayA, float p_DisplayB, const float* p_Input, float* p_Output)
{
   const int x = blockIdx.x * blockDim.x + threadIdx.x;
   const int y = blockIdx.y * blockDim.y + threadIdx.y;

   if ((x < p_Width) && (y < p_Height))
   {
       const int index = ((y * p_Width) + x) * 4;
       
       float e = 2.718281828459045;
       float pie = 3.141592653589793;
       
	   float width = p_Width;
       float height = p_Height;    	
            	  
	   float Red = p_Input[index + 0];
	   float Green = p_Input[index + 1];
	   float Blue = p_Input[index + 2];
	   
	   float expR = Red + p_ExpR/100.0f;
	   float expG = Green + p_ExpG/100.0f;
	   float expB = Blue + p_ExpB/100.0f;
	   
	   float expr1 = (p_ShadP / 2.0f) - (1.0f - p_HighP)/4.0f;
	   float expr2 = (1.0f - (1.0f - p_HighP)/2.0f) + (p_ShadP / 4.0f);
	   float expr3R = (expR - expr1) / (expr2 - expr1);
	   float expr3G = (expG - expr1) / (expr2 - expr1);
	   float expr3B = (expB - expr1) / (expr2 - expr1);
	   float expr4 =  p_ContP < 0.5f ? 0.5f - (0.5f - p_ContP)/2.0f : 0.5f + (p_ContP - 0.5f)/2.0f;
	   float expr5R = expr3R > expr4 ? (expr3R - expr4) / (2.0f - 2.0f*expr4) + 0.5f : expr3R /(2.0f*expr4);
	   float expr5G = expr3G > expr4 ? (expr3G - expr4) / (2.0f - 2.0f*expr4) + 0.5f : expr3G /(2.0f*expr4);
	   float expr5B = expr3B > expr4 ? (expr3B - expr4) / (2.0f - 2.0f*expr4) + 0.5f : expr3B /(2.0f*expr4);
	   float expr6R = (((sin(2.0f * pie * (expr5R -1.0f/4.0f)) + 1.0f) / 20.0f) * p_MidR*4.0f) + expr3R;
	   float expr6G = (((sin(2.0f * pie * (expr5G -1.0f/4.0f)) + 1.0f) / 20.0f) * p_MidG*4.0f) + expr3G;
	   float expr6B = (((sin(2.0f * pie * (expr5B -1.0f/4.0f)) + 1.0f) / 20.0f) * p_MidB*4.0f) + expr3B;
	   float midR = expR >= expr1 && expR <= expr2 ? expr6R * (expr2 - expr1) + expr1 : expR;
	   float midG = expG >= expr1 && expG <= expr2 ? expr6G * (expr2 - expr1) + expr1 : expG;
	   float midB = expB >= expr1 && expB <= expr2 ? expr6B * (expr2 - expr1) + expr1 : expB;
		
	   float shadupR1 = midR > 0.0f ? 2.0f * (midR/p_ShadP) - log((midR/p_ShadP) * (e * p_ShadR * 2.0f) + 1.0f)/log(e * p_ShadR * 2.0f + 1.0f) : midR;
	   float shadupR = midR < p_ShadP && p_ShadR > 0.0f ? (shadupR1 + p_ShadR * (1.0f - shadupR1)) * p_ShadP : midR;
	   float shadupG1 = midG > 0.0f ? 2.0f * (midG/p_ShadP) - log((midG/p_ShadP) * (e * p_ShadG * 2.0f) + 1.0f)/log(e * p_ShadG * 2.0f + 1.0f) : midG;
	   float shadupG = midG < p_ShadP && p_ShadG > 0.0f ? (shadupG1 + p_ShadG * (1.0f - shadupG1)) * p_ShadP : midG;
	   float shadupB1 = midB > 0.0f ? 2.0f * (midB/p_ShadP) - log((midB/p_ShadP) * (e * p_ShadB * 2.0f) + 1.0f)/log(e * p_ShadB * 2.0f + 1.0f) : midB;
	   float shadupB = midB < p_ShadP && p_ShadB > 0.0f ? (shadupB1 + p_ShadB * (1.0f - shadupB1)) * p_ShadP : midB;
	   
	   float shaddownR1 = shadupR/p_ShadP + p_ShadR*2 * (1.0f - shadupR/p_ShadP);
	   float shaddownR = shadupR < p_ShadP && p_ShadR < 0.0f ? (shaddownR1 >= 0.0f ? log(shaddownR1 * (e * p_ShadR * -2.0f) + 1.0f)/log(e * p_ShadR * -2.0f + 1.0f) : shaddownR1) * p_ShadP : shadupR;
	   float shaddownG1 = shadupG/p_ShadP + p_ShadG*2 * (1.0f - shadupG/p_ShadP);
	   float shaddownG = shadupG < p_ShadP && p_ShadG < 0.0f ? (shaddownG1 >= 0.0f ? log(shaddownG1 * (e * p_ShadG * -2.0f) + 1.0f)/log(e * p_ShadG * -2.0f + 1.0f) : shaddownG1) * p_ShadP : shadupG;
	   float shaddownB1 = shadupB/p_ShadP + p_ShadB*2 * (1.0f - shadupB/p_ShadP);
	   float shaddownB = shadupB < p_ShadP && p_ShadB < 0.0f ? (shaddownB1 >= 0.0f ? log(shaddownB1 * (e * p_ShadB * -2.0f) + 1.0f)/log(e * p_ShadB * -2.0f + 1.0f) : shaddownB1) * p_ShadP : shadupB;
	   
	   float highupR1 = ((shaddownR - p_HighP) / (1.0f - p_HighP)) * (1.0f + (p_HighR * 2.0f));
	   float highupR = shaddownR > p_HighP && p_HighP < 1.0f && p_HighR > 0.0f ? (2.0f * highupR1 - log(highupR1 * e * p_HighR + 1.0f)/log(e * p_HighR + 1.0f)) * (1.0f - p_HighP) + p_HighP : shaddownR;
	   float highupG1 = ((shaddownG - p_HighP) / (1.0f - p_HighP)) * (1.0f + (p_HighG * 2.0f));
	   float highupG = shaddownG > p_HighP && p_HighP < 1.0f && p_HighG > 0.0f ? (2.0f * highupG1 - log(highupG1 * e * p_HighG + 1.0f)/log(e * p_HighG + 1.0f)) * (1.0f - p_HighP) + p_HighP : shaddownG;
	   float highupB1 = ((shaddownB - p_HighP) / (1.0f - p_HighP)) * (1.0f + (p_HighB * 2.0f));
	   float highupB = shaddownB > p_HighP && p_HighP < 1.0f && p_HighB > 0.0f ? (2.0f * highupB1 - log(highupB1 * e * p_HighB + 1.0f)/log(e * p_HighB + 1.0f)) * (1.0f - p_HighP) + p_HighP : shaddownB;
	   
	   float highdownR1 = (highupR - p_HighP) / (1.0f - p_HighP);
	   float highdownR = highupR > p_HighP && p_HighP < 1.0f && p_HighR < 0.0f ? log(highdownR1 * (e * p_HighR * -2.0f) + 1.0f)/log(e * p_HighR * -2.0f + 1.0f) * (1.0f + p_HighR) * (1.0f - p_HighP) + p_HighP : highupR;
	   float highdownG1 = (highupG - p_HighP) / (1.0f - p_HighP);
	   float highdownG = highupG > p_HighP && p_HighP < 1.0f && p_HighG < 0.0f ? log(highdownG1 * (e * p_HighG * -2.0f) + 1.0f)/log(e * p_HighG * -2.0f + 1.0f) * (1.0f + p_HighG) * (1.0f - p_HighP) + p_HighP : highupG;
	   float highdownB1 = (highupB - p_HighP) / (1.0f - p_HighP);
	   float highdownB = highupB > p_HighP && p_HighP < 1.0f && p_HighB < 0.0f ? log(highdownB1 * (e * p_HighB * -2.0f) + 1.0f)/log(e * p_HighB * -2.0f + 1.0f) * (1.0f + p_HighB) * (1.0f - p_HighP) + p_HighP : highupB;
	   
	   float contR = (highdownR - p_ContP) * p_ContR + p_ContP;
	   float contG = (highdownG - p_ContP) * p_ContG + p_ContP;
	   float contB = (highdownB - p_ContP) * p_ContB + p_ContP;
	   
	   float luma = contR * 0.2126f + contG * 0.7152f + contB * 0.0722f;
	   float satR = (1.0f - (p_SatR*0.2126f + p_SatG* 0.7152f + p_SatB*0.0722f)) * luma + contR * p_SatR;
	   float satG = (1.0f - (p_SatR*0.2126f + p_SatG* 0.7152f + p_SatB*0.0722f)) * luma + contG * p_SatG;
	   float satB = (1.0f - (p_SatR*0.2126f + p_SatG* 0.7152f + p_SatB*0.0722f)) * luma + contB * p_SatB;
	   
	   float DexpR = (x / width) + p_ExpR/100.0f;
	   float DexpG = (x / width) + p_ExpG/100.0f;
	   float DexpB = (x / width) + p_ExpB/100.0f;
		
	   float Dexpr1 = (p_ShadP / 2.0f) - (1.0f - p_HighP)/4.0f;
	   float Dexpr2 = (1.0f - (1.0f - p_HighP)/2.0f) + (p_ShadP / 4.0f);
	   float Dexpr3R = (DexpR - Dexpr1) / (Dexpr2 - Dexpr1);
	   float Dexpr3G = (DexpG - Dexpr1) / (Dexpr2 - Dexpr1);
	   float Dexpr3B = (DexpB - Dexpr1) / (Dexpr2 - Dexpr1);
	   float Dexpr4 =  p_ContP < 0.5f ? 0.5f - (0.5f - p_ContP)/2.0f : 0.5f + (p_ContP - 0.5f)/2.0f;
	   float Dexpr5R = Dexpr3R > Dexpr4 ? (Dexpr3R - Dexpr4) / (2.0f - 2.0f*Dexpr4) + 0.5f : Dexpr3R /(2.0f*Dexpr4);
	   float Dexpr5G = Dexpr3G > Dexpr4 ? (Dexpr3G - Dexpr4) / (2.0f - 2.0f*Dexpr4) + 0.5f : Dexpr3G /(2.0f*Dexpr4);
	   float Dexpr5B = Dexpr3B > Dexpr4 ? (Dexpr3B - Dexpr4) / (2.0f - 2.0f*Dexpr4) + 0.5f : Dexpr3B /(2.0f*Dexpr4);
	   float Dexpr6R = (((sin(2.0f * pie * (Dexpr5R -1.0f/4.0f)) + 1.0f) / 20.0f) * p_MidR*4.0f) + Dexpr3R;
	   float Dexpr6G = (((sin(2.0f * pie * (Dexpr5G -1.0f/4.0f)) + 1.0f) / 20.0f) * p_MidG*4.0f) + Dexpr3G;
	   float Dexpr6B = (((sin(2.0f * pie * (Dexpr5B -1.0f/4.0f)) + 1.0f) / 20.0f) * p_MidB*4.0f) + Dexpr3B;
	   float DmidR = DexpR >= Dexpr1 && DexpR <= Dexpr2 ? Dexpr6R * (Dexpr2 - Dexpr1) + Dexpr1 : DexpR;
	   float DmidG = DexpG >= Dexpr1 && DexpG <= Dexpr2 ? Dexpr6G * (Dexpr2 - Dexpr1) + Dexpr1 : DexpG;
	   float DmidB = DexpB >= Dexpr1 && DexpB <= Dexpr2 ? Dexpr6B * (Dexpr2 - Dexpr1) + Dexpr1 : DexpB;
	   
	   float DshadupR1 = DmidR > 0.0f ? 2.0f * (DmidR/p_ShadP) - log((DmidR/p_ShadP) * (e * p_ShadR * 2.0f) + 1.0f)/log(e * p_ShadR * 2.0f + 1.0f) : DmidR;
	   float DshadupR = DmidR < p_ShadP && p_ShadR > 0.0f ? (DshadupR1 + p_ShadR * (1.0f - DshadupR1)) * p_ShadP : DmidR;
	   float DshadupG1 = DmidG > 0.0f ? 2.0f * (DmidG/p_ShadP) - log((DmidG/p_ShadP) * (e * p_ShadG * 2.0f) + 1.0f)/log(e * p_ShadG * 2.0f + 1.0f) : DmidG;
	   float DshadupG = DmidG < p_ShadP && p_ShadG > 0.0f ? (DshadupG1 + p_ShadG * (1.0f - DshadupG1)) * p_ShadP : DmidG;
	   float DshadupB1 = DmidB > 0.0f ? 2.0f * (DmidB/p_ShadP) - log((DmidB/p_ShadP) * (e * p_ShadB * 2.0f) + 1.0f)/log(e * p_ShadB * 2.0f + 1.0f) : DmidB;
	   float DshadupB = DmidB < p_ShadP && p_ShadB > 0.0f ? (DshadupB1 + p_ShadB * (1.0f - DshadupB1)) * p_ShadP : DmidB;
	   
	   float DshaddownR1 = (DshadupR/p_ShadP) + (p_ShadR * 2.0f * (1.0f - DshadupR/p_ShadP));
	   float DshaddownR = DshadupR < p_ShadP && p_ShadR < 0.0f ? (DshaddownR1 >= 0.0f ? log(DshaddownR1 * (e * p_ShadR * -2.0f) + 1.0f)/log(e * p_ShadR * -2.0f + 1.0f) : DshaddownR1) * p_ShadP : DshadupR;
	   float DshaddownG1 = (DshadupG/p_ShadP) + (p_ShadG * 2.0f * (1.0f - DshadupG/p_ShadP));
	   float DshaddownG = DshadupG < p_ShadP && p_ShadG < 0.0f ? (DshaddownG1 >= 0.0f ? log(DshaddownG1 * (e * p_ShadG * -2.0f) + 1.0f)/log(e * p_ShadG * -2.0f + 1.0f) : DshaddownG1) * p_ShadP : DshadupG;
	   float DshaddownB1 = (DshadupB/p_ShadP) + (p_ShadB * 2.0f * (1.0f - DshadupB/p_ShadP));
	   float DshaddownB = DshadupB < p_ShadP && p_ShadB < 0.0f ? (DshaddownB1 >= 0.0f ? log(DshaddownB1 * (e * p_ShadB * -2.0f) + 1.0f)/log(e * p_ShadB * -2.0f + 1.0f) : DshaddownB1) * p_ShadP : DshadupB;
	   
	   float DhighupR1 = ((DshaddownR - p_HighP) / (1.0f - p_HighP)) * (1.0f + (p_HighR * 2.0f));
	   float DhighupR = DshaddownR > p_HighP && p_HighP < 1.0f && p_HighR > 0.0f ? (2.0f * DhighupR1 - log(DhighupR1 * e * p_HighR + 1.0f)/log(e * p_HighR + 1.0f)) * (1.0f - p_HighP) + p_HighP : DshaddownR;
	   float DhighupG1 = ((DshaddownG - p_HighP) / (1.0f - p_HighP)) * (1.0f + (p_HighG * 2.0f));
	   float DhighupG = DshaddownG > p_HighP && p_HighP < 1.0f && p_HighG > 0.0f ? (2.0f * DhighupG1 - log(DhighupG1 * e * p_HighG + 1.0f)/log(e * p_HighG + 1.0f)) * (1.0f - p_HighP) + p_HighP : DshaddownG;
	   float DhighupB1 = ((DshaddownB - p_HighP) / (1.0f - p_HighP)) * (1.0f + (p_HighB * 2.0f));
	   float DhighupB = DshaddownB > p_HighP && p_HighP < 1.0f && p_HighB > 0.0f ? (2.0f * DhighupB1 - log(DhighupB1 * e * p_HighB + 1.0f)/log(e * p_HighB + 1.0f)) * (1.0f - p_HighP) + p_HighP : DshaddownB;
	   
	   float DhighdownR1 = (DhighupR - p_HighP) / (1.0f - p_HighP);
	   float DhighdownR = DhighupR > p_HighP && p_HighP < 1.0f && p_HighR < 0.0f ? log(DhighdownR1 * (e * p_HighR * -2.0f) + 1.0f)/log(e * p_HighR * -2.0f + 1.0f) * (1.0f + p_HighR) * (1.0f - p_HighP) + p_HighP  : DhighupR;
	   float DhighdownG1 = (DhighupG - p_HighP) / (1.0f - p_HighP);
	   float DhighdownG = DhighupG > p_HighP && p_HighP < 1.0f && p_HighG < 0.0f ? log(DhighdownG1 * (e * p_HighG * -2.0f) + 1.0f)/log(e * p_HighG * -2.0f + 1.0f) * (1.0f + p_HighG) * (1.0f - p_HighP) + p_HighP  : DhighupG;
	   float DhighdownB1 = (DhighupB - p_HighP) / (1.0f - p_HighP);
	   float DhighdownB = DhighupB > p_HighP && p_HighP < 1.0f && p_HighB < 0.0f ? log(DhighdownB1 * (e * p_HighB * -2.0f) + 1.0f)/log(e * p_HighB * -2.0f + 1.0f) * (1.0f + p_HighB) * (1.0f - p_HighP) + p_HighP  : DhighupB;
	   
	   float DcontR = (DhighdownR - p_ContP) * p_ContR + p_ContP;
	   float DcontG = (DhighdownG - p_ContP) * p_ContG + p_ContP;
	   float DcontB = (DhighdownB - p_ContP) * p_ContB + p_ContP;
	   
	   float Dluma = DcontR * 0.2126f + DcontG * 0.7152f + DcontB * 0.0722f;
	   float DsatR = (1.0f - (p_SatR*0.2126f + p_SatG* 0.7152f + p_SatB*0.0722f)) * Dluma + DcontR * p_SatR;
	   float DsatG = (1.0f - (p_SatR*0.2126f + p_SatG* 0.7152f + p_SatB*0.0722f)) * Dluma + DcontG * p_SatG;
	   float DsatB = (1.0f - (p_SatR*0.2126f + p_SatG* 0.7152f + p_SatB*0.0722f)) * Dluma + DcontB * p_SatB;
	   
	   float overlayR = y/(height) >= p_ShadP && y/(height) <= p_ShadP + 0.005f ? (fmodf(x, 2.0f) != 0.0f ? 1.0f : 0.0f) : DsatR >= (y - 5)/(height) && DsatR <= (y + 5)/(height) ? 1.0f : 0.0f;
	   float overlayG = y/(height) >= p_HighP && y/(height) <= p_HighP + 0.005f ? (fmodf(x, 2.0f) != 0.0f ? 1.0f : 0.0f) : DsatG >= (y - 5)/(height) && DsatG <= (y + 5)/(height) ? 1.0f : 0.0f;
	   float overlayB = y/(height) >= p_ContP && y/(height) <= p_ContP + 0.005f ? (fmodf(x, 2.0f) != 0.0f ? 1.0f : 0.0f) : DsatB >= (y - 5)/(height) && DsatB <= (y + 5)/(height) ? 1.0f : 0.0f;
	   
	   float outR = p_DisplayA == 1.0f && p_DisplayB == 1.0f ? (overlayR == 0.0f ? satR : overlayR) : p_DisplayA == 1.0f ? overlayR : satR;
       float outG = p_DisplayA == 1.0f && p_DisplayB == 1.0f ? (overlayG == 0.0f ? satG : overlayG) : p_DisplayA == 1.0f ? overlayG : satG;
       float outB = p_DisplayA == 1.0f && p_DisplayB == 1.0f ? (overlayB == 0.0f ? satB : overlayB) : p_DisplayA == 1.0f ? overlayB : satB;
				 			
       p_Output[index + 0] = outR;
       p_Output[index + 1] = outG;
       p_Output[index + 2] = outB;
       p_Output[index + 3] = p_Input[index + 3];
   }
}

void RunCudaKernel(int p_Width, int p_Height, float* p_Exp, float* p_Cont, float* p_Sat, 
float* p_Shad, float* p_Mid, float* p_High, float* p_Pivot, float* p_Display, const float* p_Input, float* p_Output)
{
    dim3 threads(128, 1, 1);
    dim3 blocks(((p_Width + threads.x - 1) / threads.x), p_Height, 1);

    FilmGradeKernel<<<blocks, threads>>>(p_Width, p_Height, p_Exp[0], p_Exp[1], p_Exp[2], p_Cont[0], p_Cont[1], p_Cont[2], 
    p_Sat[0], p_Sat[1], p_Sat[2], p_Shad[0], p_Shad[1], p_Shad[2], p_Mid[0], p_Mid[1], p_Mid[2], p_High[0], p_High[1], p_High[2], 
    p_Pivot[0], p_Pivot[1], p_Pivot[2], p_Display[0], p_Display[1], p_Input, p_Output);
}
